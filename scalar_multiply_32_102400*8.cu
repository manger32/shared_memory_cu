/*В данном задании требуется представить 2 варианта программы для видеокарты: 1) максимально простой и короткий; и 2) быстрый, использующий разделяемую память.
Запрограммируйте генерацию случайных входных данных для алгоритма и автоматическую проверку корректности работы программы.
Выполните теоретическую оценку производительности обоих вариантов алгоритма. Укажите в отчете, насколько теоретическая оценка отличается от практической. */

/*Реализуйте умножение длинной матрицы, хранящейся по столбцам, на длинный вектор*/

#include <hip/hip_runtime.h>
#include <iostream>
#define N 32 //shortest dimension of A: 1536 ; for test 50
#define M (102400*8) //*100 or 10000

using namespace std;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

__global__ void Multiply(int *A, int *B, int *C){
// calculate the row & col index of the element
    int row = blockIdx.x*blockDim.x + threadIdx.x;
    if (row >= N)
      return;
    int result = 0;
// product between row of a and b
    for(int k = 0; k < M; ++k)
    {
        result += A[row + k*N] * B[k];
        //printf("%d ", result);
    }
    C[row] = result;
}

int main(int argc, char **argv)
{
  srand(time(NULL));
  int *A = new int [N*M];
  int *b = new int [M];
  int *res_CPU = new int[N];
  int *res_GPU = new int[N];
  int i, j;
  for(i = 0; i < N; ++i)
      res_CPU[i] = 0;
  for(i = 0; i < N; ++i)
  {
      for(j = 0; j < M; ++j)
      {
          A[i + j*N] = rand() % 10;
          //cout << A[i*N + j] << " ";
      }
      //cout << endl;
  }
  //cout << endl;
  for(i = 0; i < M; ++i)
  {
      b[i] = rand() % 10;
      //cout << b[i] << " ";
  }
  //cout << endl;
  // shared memory: t = 0..32 - warp
  clock_t startCPU = clock();
  for(i = 0; i < N; ++i)
  {
      for(j = 0; j < M; ++j)
          res_CPU[i] += A[i + j*N]*b[j];
      //cout << "Res_CPU[" << i << "] = " << res_CPU[i] << " " << endl;
  }
  double elapsedTimeCPU = (double)(clock()-startCPU)/CLOCKS_PER_SEC;
  cout << "CPU product time = " << elapsedTimeCPU*1000 << " ms\n";

  int (*aA), (*aB), (*aRes);
  hipEvent_t startCUDA, stopCUDA;
  float elapsedTimeCUDA;
  hipEventCreate(&startCUDA);
  hipEventCreate(&stopCUDA);

  CHECK(hipMalloc((void**)&aA, (N*M)*sizeof(int)));
  CHECK(hipMalloc((void**)&aB, (M)*sizeof(int)));
  CHECK(hipMalloc((void**)&aRes, (N)*sizeof(int)));

  CHECK(hipMemcpy(aA, A, (N*M)*sizeof(int), hipMemcpyHostToDevice));
  CHECK(hipMemcpy(aB, b, (M)*sizeof(int), hipMemcpyHostToDevice));
  //CHECK(cudaMemcpy(aRes, res_CPU, (N)*sizeof(int), cudaMemcpyHostToDevice));

  //int numBlocks = 1;
  //dim3 threadsPerBlock(N,N);
  hipEventRecord(startCUDA,0);
  Multiply<<<(N+511)/512, 512>>>(aA,aB,aRes);
  hipEventRecord(stopCUDA,0);
  hipEventSynchronize(stopCUDA);
  CHECK(hipGetLastError());
  CHECK(hipMemcpy(res_GPU, aRes, N*sizeof(int), hipMemcpyDeviceToHost));

  hipEventElapsedTime(&elapsedTimeCUDA, startCUDA, stopCUDA);

  cout << "CUDA product time = " << elapsedTimeCUDA << " ms\n";
  cout << "CUDA memory throughput = " << 3*N*sizeof(float)/elapsedTimeCUDA/1024/1024/1.024 << " Gb/s\n";
  /*for (i = 0; i < N; i++) {
    cout << "Res_GPU[" << i << "] = " << res_GPU[i] << " " << endl;
  }*/
  for (i = 0; i < N; i++) {
    if (res_CPU[i] != res_GPU[i])
    {
      cout << "Not equal. Try again, again." << endl;
      break;
    }
  }
  CHECK(hipFree(aA));
  CHECK(hipFree(aB));
  CHECK(hipFree(aRes));
  return 0;
}
